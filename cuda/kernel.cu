#include "hip/hip_runtime.h"

#include <stdio.h>
#include <chrono>

__global__ void fizzBuzzKernel(int *matrix, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = row * n + col + 1; // Compute the number for this thread

    if (tid <= n * n) {
        if (tid % 15 == 0) {
            matrix[row * n + col] = 0; // 0 indicates "FizzBuzz"
        } else if (tid % 3 == 0) {
            matrix[row * n + col] = 1; // 1 indicates "Fizz"
        } else if (tid % 5 == 0) {
            matrix[row * n + col] = 2; // 2 indicates "Buzz"
        } else {
            matrix[row * n + col] = tid; // Number itself
        }
    }
}

int main() {
    const int n = 10000; // Set the size of the matrix (n x n)

    // Allocate memory on the host for the matrix
    int *host_matrix = (int*)malloc(n * n * sizeof(int));

    // Allocate memory on the device for the matrix
    int *device_matrix;
    hipMalloc((void**)&device_matrix, n * n * sizeof(int));

    auto start_time = std::chrono::high_resolution_clock::now();

    // Launch the CUDA kernel with 2D grid and block dimensions
    dim3 block_dim(32, 32);
    dim3 grid_dim((n + block_dim.x - 1) / block_dim.x, (n + block_dim.y - 1) / block_dim.y);
    fizzBuzzKernel<<<grid_dim, block_dim>>>(device_matrix, n);
    auto end_time = std::chrono::high_resolution_clock::now();

    // Copy the matrix from the device to the host
    hipMemcpy(host_matrix, device_matrix, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // Display the results
    for (int i = 1; i < 10; i++) {
        if (host_matrix[i] == 0) {
            printf("FizzBuzz\n");
        } else if (host_matrix[i] == 1) {
            printf("Fizz\n");
        } else if (host_matrix[i] == 2) {
            printf("Buzz\n");
        } else {
            printf("%d\n", host_matrix[i]);
        }
    }

    auto duration_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count();

    printf("Execution time ns: %lld\n", duration_ns);

    // Free device and host memory
    hipFree(device_matrix);
    free(host_matrix);

    return 0;
}